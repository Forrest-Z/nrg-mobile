#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 256

__device__ bool d_error;

__global__ void crs(float* fPoints, int* iPoints, int cloudSize, int maxRay, float* fStart, int* iStart, float* fStartVoxel, int* outPoints, int* outSizes, int* min, int* max, float resolution)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  // if there are extra threads, don't run them
  if(index >= cloudSize)
    return;

  // init vars
  bool done = false;

  // calculate normal vector in direction of sensor->point
  float direction[3] = {fPoints[index]-fStart[0], fPoints[cloudSize+index]-fStart[1], fPoints[2*cloudSize+index]-fStart[2]};
  float directionMagnitude = powf(powf(direction[0],2) + powf(direction[1],2) + powf(direction[2],2),0.5);

  // variables used for ray casting algorithm
  int stepDirection[3];        // +/- step in each cardinal direction
  float accumulatedError[3];   // error accumulated in each direction
  float deltaError[3];         // change in error accumulated for a step in a direction
  int currentIndex[3];         // for tracking the index as we trace
  int pointIndex[3];           // index of final occupied point
  bool usePI = true;          // we only check for the final point if it is on the map, 
                               // otherwise we are done when we leave the map

  // Set the starting position to the sensor position, and the final index
  for(int i = 0; i < 3; i++)
  {
    currentIndex[i] = iStart[i];
  }
  pointIndex[0] = iPoints[index];
  pointIndex[1] = iPoints[cloudSize+index];
  pointIndex[2] = iPoints[2*cloudSize+index];

  // If the occupied point is in the map, we use it as a stopping point
  if(pointIndex[0] < min[0] || pointIndex[0] > max[0] ||
     pointIndex[1] < min[1] || pointIndex[1] > max[1] ||
     pointIndex[2] < min[2] || pointIndex[2] > max[2])
    usePI = false;

  // check direction magnitude for divide by zero or same cell
  if(fabs(directionMagnitude) < resolution)
  {
    d_error = true;
    return;
  }

  // set up initial values in each direction
  for(int dir = 0; dir < 3; dir++)
  {
    direction[dir] = fdividef(direction[dir],directionMagnitude);
    if(direction[dir] > 0.0)
      stepDirection[dir] = 1;
    else if(direction[dir] < 0.0)
      stepDirection[dir] = -1;

    float voxelBorder = fStartVoxel[dir] + stepDirection[dir]*resolution*0.5;
    accumulatedError[dir] = fdividef((voxelBorder - fStart[dir]),direction[dir]);
    deltaError[dir] = fdividef(resolution,fabs(direction[dir]));
  }
  
  int count = 0;
  // loop until we are out of map bounds
  while(!done)
  {
    // find direction of min error
    int dim = 2;
    if(fabs(accumulatedError[0]) < fabs(accumulatedError[1]) && fabs(accumulatedError[0]) < fabs(accumulatedError[2]))
      dim = 0;
    else if(fabs(accumulatedError[1]) < fabs(accumulatedError[0]) && fabs(accumulatedError[1]) < fabs(accumulatedError[2]))
      dim = 1;

    // advance in direction of min error
    currentIndex[dim] = currentIndex[dim] + stepDirection[dim];
    accumulatedError[dim] = accumulatedError[dim] + deltaError[dim]; 

    // done if we are at occ point
    if(usePI)
    {
      if(currentIndex[0] == pointIndex[0] &&
         currentIndex[1] == pointIndex[1] &&
         currentIndex[2] == pointIndex[2])
      {
        done = true;
      }
    }

    // if we are off the map, we are done. 
    if(currentIndex[0] < min[0] || currentIndex[0] > max[0] ||
       currentIndex[1] < min[1] || currentIndex[1] > max[1] ||
       currentIndex[2] < min[2] || currentIndex[2] > max[2])
    {
      done = true;
    }
      
    //otherwise we mark the current index as unoccupied
    if(!done)
    {
      outPoints[index*maxRay+count] = currentIndex[0];
      outPoints[index*maxRay+(cloudSize*maxRay)+count] = currentIndex[1];
      outPoints[index*maxRay+(2*cloudSize*maxRay)+count] = currentIndex[2];
    }
    count = count + 1;
  }
  outSizes[index] = count;
  return;
}

bool castRays(float* fPoints, int* iPoints, int cloudSize, int maxRay, float* fStart, int* iStart, float* fStartVoxel, int* outPoints, int* outSizes, int minX, int maxX, int minY, int maxY, int minZ, int maxZ, float resolution) 
{
  // Device copies of three inputs and output, size of allocated memory, num of threads and blocks
  float *d_fPoints, *d_fStart, *d_fStartVoxel;
  int *d_iPoints, *d_outPoints, *d_iStart, *d_outSizes, *d_min, *d_max;
  int min[3] = {minX, minY, minZ};
  int max[3] = {maxX, maxY, maxZ};
  int thr, blk;
  bool h_error = false;
  int temp;
  for(int i = 0; i < 3; i ++)
  {
    if(min[i] > max[i])
    {
      temp = min[i];
      min[i] = max[i];
      max[i] = temp;
    }
  }
  //cudaMemset(&d_error,0,sizeof(bool));

  // Alloc memory for device copies of inputs and outputs
  hipMalloc((void**)&d_fPoints, ((cloudSize*3) * sizeof(float)));
  hipMalloc((void**)&d_iPoints, ((cloudSize*3) * sizeof(int)));
  hipMalloc((void**)&d_fStart, (3 * sizeof(float)));
  hipMalloc((void**)&d_iStart, (3 * sizeof(int)));
  hipMalloc((void**)&d_fStartVoxel, (3 * sizeof(float)));
  hipMalloc((void**)&d_min, (3 * sizeof(int)));
  hipMalloc((void**)&d_max, (3 * sizeof(int)));
  hipMalloc((void**)&d_outPoints, ((cloudSize*maxRay*3) * sizeof(int)));
  hipMalloc((void**)&d_outSizes, (cloudSize * sizeof(int)));

  // Copy inputs to device
  hipMemcpy(d_fPoints, fPoints, ((cloudSize*3) * sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(d_iPoints, iPoints, ((cloudSize*3) * sizeof(int)), hipMemcpyHostToDevice);
  hipMemcpy(d_fStart, fStart, (3 * sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(d_iStart, iStart, (3 * sizeof(int)), hipMemcpyHostToDevice);
  hipMemcpy(d_fStartVoxel, fStartVoxel, (3 * sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(d_min, min, (3 * sizeof(int)), hipMemcpyHostToDevice);
  hipMemcpy(d_max, max, (3 * sizeof(int)), hipMemcpyHostToDevice);

  // Calculates blocks and threads and launch average3 kernel on GPU
  thr=THREADS_PER_BLOCK;
  blk=cloudSize/THREADS_PER_BLOCK+1;
  crs<<<blk,thr>>>(d_fPoints, d_iPoints, cloudSize, maxRay, d_fStart, d_iStart, d_fStartVoxel, d_outPoints, 
                   d_outSizes, d_min, d_max, resolution);

  // Wait for the GPU to finish
  hipDeviceSynchronize();

  //// Copy result back to host and cleanup
  hipMemcpy(outPoints, d_outPoints, (cloudSize*maxRay*3) * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(outSizes, d_outSizes, cloudSize * sizeof(int), hipMemcpyDeviceToHost);
  //cudaMemcpyFromSymbol(&h_error, "d_error", sizeof(bool), 0, cudaMemcpyDeviceToHost);
  hipFree(d_outSizes); 
  hipFree(d_outPoints);  
  hipFree(d_max);  
  hipFree(d_min);  
  hipFree(d_fStartVoxel); 
  hipFree(d_iStart); 
  hipFree(d_fStart); 
  hipFree(d_iPoints); 
  hipFree(d_fPoints);
  return !h_error;
}
